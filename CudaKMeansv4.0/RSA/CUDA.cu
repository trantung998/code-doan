#include "hip/hip_runtime.h"
/***************************************
 * CUDA.cu
 * Author: Andrei Kopytov
 * Date of creation: Oct 12, 2012
 * Last modified: Oct 17, 2012
 **************************************/
 
#include "CUDA.h"
#include <iostream>
using std::cout;
using std::endl;

// Static members
int		CUDA::m_numDevices = 0;
int 	CUDA::m_activeDeviceID = 0;
bool 	CUDA::m_cudaSupported = 0;
bool 	CUDA::m_cudaInitialized = 0;
int 	CUDA::m_maxThreadsPerMultiProcessor = 0;

__global__ void kernel()
{
	// TODO: parallelize BigInt's longMultiply() function
}

bool CUDA::checkForCUDASupport()
{
	hipError_t error;
	
	error = hipGetDeviceCount(&m_numDevices);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
	if (!m_numDevices) {
		cout << "No CUDA capable devices found!" << endl;
		return false;
	}
	
	return true;
}

void CUDA::initializeCUDA()
{
	hipError_t error;
	m_cudaInitialized = false;
	
	m_cudaSupported = checkForCUDASupport();
	if (!m_cudaSupported) {
		cout << "CUDA could not be initialized properly. Defaulting to serial processing!" << endl;
		return;
	}
	
	/* We're going to choose the first available CUDA device. In the future this can be modified
	 * to go through all available CUDA devices and choose the best/most capable one, or to utilize all
	 * available devices for parallel processing.
	 */
	error = hipGetDevice(&m_activeDeviceID);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
		return;
	}
	hipDeviceProp_t properties;
	error = hipGetDeviceProperties(&properties, m_activeDeviceID);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
		return;
	}
	
	m_maxThreadsPerMultiProcessor = properties.maxThreadsPerMultiProcessor;
	m_cudaInitialized = true;
}

bool CUDA::isSupported()
{
	return m_cudaSupported;
}

bool CUDA::isInitialized()
{
	return m_cudaInitialized;
}

void CUDA::launchGrid(unsigned long int na, unsigned long int nb, unsigned char* result)
{
	// TODO: figure out the correct data types and allocate memory for them
}