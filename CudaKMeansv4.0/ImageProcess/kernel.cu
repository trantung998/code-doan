#include "hip/hip_runtime.h"
#include "CImg.h"
#include <ctime>
#include <iostream>
#include "filter.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_functions.h>
#include <hip/device_functions.h>
#include <>
#include <math_functions.h>


typedef struct{
	UCHAR *r;
	UCHAR *g;
	UCHAR *b;
}imageData;

__constant__ float cst_ptr [9];

using namespace std;
using namespace cimg_library;
#define PI 3.14

__global__ void gray_kernel(UCHAR *red, UCHAR *green, UCHAR *blue
					 ,UCHAR *gray_image, UINT w, UINT h
					 )
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;
	int index = tx*h+ty;

	if(index< w*h)
		gray_image[index] = round(0.299*((double)red[index])+ 0.587*((double)green[index]) + 0.114*((double)blue[index]));
}
__global__ void filter(imageData scr, imageData des, float *_filter, 
					   UINT w, UINT h, UINT filterSIZE, 
					   float factor, float bias)
{
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int ty = blockDim.y * blockIdx.y + threadIdx.y;
	int index = tx*h+ty;
	if(index < w*h)
	{
		UCHAR r = 0, g = 0, b = 0;

		for(int i = 0; i< filterSIZE; i++)
		{
			for(int j = 0; j < filterSIZE ; j++)
			{
				int imgX = (tx - filterSIZE/2 + i + w)% w;
				int imgY = (ty - filterSIZE/2 + i + h)% h;
				r += scr.r[imgX*h + imgY]* _filter[i*filterSIZE + j];
				g += scr.g[imgX*h + imgY]* _filter[i*filterSIZE + j];
				b += scr.b[imgX*h + imgY]* _filter[i*filterSIZE + j];
			}
		}

		des.r[index] = min(max(int(factor * r + bias), 0), 255); 
		des.g[index] = min(max(int(factor * g + bias), 0), 255); 
		des.b[index] = min(max(int(factor * b + bias), 0), 255); 
	}
}
void call_kernel(imageData data,UINT w,UINT h)
{
	//hipMemcpyToSymbol(HIP_SYMBOL(cst_ptr), host_ptr, data_size );
	UCHAR *dr, *dg, *db;
	checkCudaErrors(hipMalloc((void**)&dr, w*h));
	checkCudaErrors(hipMalloc((void**)&dg, w*h));
	checkCudaErrors(hipMalloc((void**)&db, w*h));

	checkCudaErrors(hipMemcpy(dr,data.r,w*h,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dg,data.g,w*h,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(db,data.b,w*h,hipMemcpyHostToDevice));

	UCHAR *d_result, *h_result;
	checkCudaErrors(hipMalloc((void**)&d_result, w*h));
	h_result = (UCHAR*)malloc(w*h);

	dim3 numThreadPerBlock(32,32);
	dim3 grid(ceil((float)w/32),ceil((float)h/32));

	printf("numblock %d\n",grid.x);
	printf("numblock %d\n",grid.y);

	gray_kernel<<<grid,numThreadPerBlock>>>(dr,dg,db,d_result,w,h);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipMemcpy(h_result,d_result,w*h,hipMemcpyDeviceToHost));
	CImg<UCHAR> result(w,h,1,1);
	for(int i=0;i<w;i++){
		for(int j=0;j<h;j++){
			result(i,j,0,0) = h_result[i*h + j];	
		}
	}
	result.display("GrayScale");
}
void call_kernel_filter(imageData data,UINT w,UINT h)
{
	imageData d_Data, d_Result;
	
	float *d_filter;
	//Malloc memory
	checkCudaErrors(hipMalloc((void**)&d_Data.r, w*h));
	checkCudaErrors(hipMalloc((void**)&d_Data.g, w*h));
	checkCudaErrors(hipMalloc((void**)&d_Data.b, w*h));

	checkCudaErrors(hipMalloc((void**)&d_Result.r, w*h));
	checkCudaErrors(hipMalloc((void**)&d_Result.g, w*h));
	checkCudaErrors(hipMalloc((void**)&d_Result.b, w*h));

	checkCudaErrors(hipMalloc(&d_filter, 9*sizeof(float)));
	//copy data

	checkCudaErrors(hipMemcpy(d_Data.r,data.r,w*h,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Data.g,data.g,w*h,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Data.b,data.b,w*h,hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(d_filter,filter_Blur,9*sizeof(float),hipMemcpyHostToDevice));

	dim3 block(32,32);
	dim3 grid(ceil((float)w/32),ceil((float)h/32));
	float factor = 2.0;
	float bias = 0.0;

	filter<<<grid,block>>>(d_Data,d_Result,d_filter,w,h,
		3,factor,bias);

	imageData h_Result;
	 h_Result.r = (UCHAR*)malloc(w*h);
	 h_Result.g = (UCHAR*)malloc(w*h);
	 h_Result.b = (UCHAR*)malloc(w*h);

	checkCudaErrors(hipMemcpy(h_Result.r,d_Result.r,w*h,hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_Result.g,d_Result.g,w*h,hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_Result.b,d_Result.b,w*h,hipMemcpyDeviceToHost));

	CImg<UCHAR> result(w,h,1,3);
	for(int i=0;i<w;i++){
		for(int j=0;j<h;j++){
			result(i,j,0,0) = h_Result.r[i*h + j];	
			result(i,j,0,1) = h_Result.g[i*h + j];
			result(i,j,0,2) = h_Result.b[i*h + j];
		}
	}
	result.display("GrayScale");
	
}
void host_filter(imageData data, UINT w, UINT h)
{
	float factor = 1.0, bias = 0.0;
	for(int i = 0; i<)
}
int main() 
{
	CImg<UCHAR> image("hi.jpg");
	int width = image.width();
	int height = image.height();
	int depth = image.depth();
	

	CImg<UCHAR> gray1(width,height,depth,3);
	CImg<UCHAR> gray2(width,height,depth,3);
	//gray2 = image.blur(1.5);

	UCHAR *hr,*hg,*hb;
	UCHAR *result = new UCHAR[width*height];

	imageData h_image;
	h_image.r = new UCHAR[width*height];
	h_image.g = new UCHAR[width*height];
	h_image.b = new UCHAR[width*height];

	printf("Load data\n");

	for(int i=0;i<width;i++){
		for(int j=0;j<height;j++){
			h_image.r[i*height +j] = image(i,j,0,0); // RED
			h_image.g[i*height +j] = image(i,j,0,1); // GREEN
			h_image.b[i*height +j] = image(i,j,0,2); // BLUE
			//printf("%d \n",h_image.r[i*height +j]);
			//Y = 0.299*R + 0.587*G + 0.114*B
			//result[i*height +j] = round(0.299*((double)h_image.r[i*height +j]) + 0.587*((double)h_image.g[i*height +j]) + 0.114*((double)h_image.b[i*height +j]));
			//gray1(i,j,0,0) = (UCHAR)h_image.r[i*height +j];	
			//gray1(i,j,0,1) = (UCHAR)h_image.g[i*height +j];	
			//gray1(i,j,0,2) = (UCHAR)h_image.b[i*height +j]	;
		}
	}
	printf("size of h_image: %d\n",h_image.b[100]);
	getchar();
	clock_t begin = clock();	
	double elapsed_secs_1 = double(clock() - begin);	
	std::cout << "elapsed time:  " << elapsed_secs_1 << " msecs" << std::endl; 
	//call_kernel(h_image,width,height);
	call_kernel_filter(h_image,width,height);
	//gray1.save("gray1.bmp");
	//gray2.save("gray2.bmp");
 
	//show all images

	//(image,gray1).display("original");

  return 0;
}